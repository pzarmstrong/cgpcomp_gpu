#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "stdafx.h"
#include <stdio.h>
#include <string>
#include <iostream>
#include <fstream>

#include "kernel.cuh"
#include "RunTimer.h"

using namespace std;

RunTimer rt;
RunTimer execute;

__device__ bool getPosKernel(int x, int y, bool *d_pixBinaryMap, int d_width)
{
	return d_pixBinaryMap[x + y*d_width];
}

__device__ int findHazardKernel(int x, int y, int x_delta, int y_delta, bool *d_pixBinaryMap, int d_width, int d_height)
{
	int count = 0;
	do
	{
		x += x_delta;
		y += y_delta;
		count++;
	} while (0 <= x && x <= d_width && 0 <= y && y <= d_height && !getPosKernel(x, y, d_pixBinaryMap, d_width));
	return count - 1;
}

__global__ void kernel(bool *d_pixBinaryMap, int *d_pixHazardMap, int d_width, int d_height)
{
	const int compassDirections = 16;
	int directions[compassDirections];
	int min_element;

	int x = threadIdx.x + blockDim.x * blockIdx.x;
	int y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x < d_width && y < d_height)
	{
		directions[0] = findHazardKernel(x, y, 0, -1, d_pixBinaryMap, d_width, d_height);		// N

		directions[1] = findHazardKernel(x, y, 1, -2, d_pixBinaryMap, d_width, d_height);		// NNE
		directions[2] = findHazardKernel(x, y, 1, -1, d_pixBinaryMap, d_width, d_height);		// NE
		directions[3] = findHazardKernel(x, y, 2, -1, d_pixBinaryMap, d_width, d_height);		// ENE

		directions[4] = findHazardKernel(x, y, 1, 0, d_pixBinaryMap, d_width, d_height);		// E

		directions[5] = findHazardKernel(x, y, 2, 1, d_pixBinaryMap, d_width, d_height);		// ESE
		directions[6] = findHazardKernel(x, y, 1, 1, d_pixBinaryMap, d_width, d_height);		// SE
		directions[7] = findHazardKernel(x, y, 1, 2, d_pixBinaryMap, d_width, d_height);		// SSE

		directions[8] = findHazardKernel(x, y, 0, 1, d_pixBinaryMap, d_width, d_height);		// S

		directions[9] = findHazardKernel(x, y, -1, 2, d_pixBinaryMap, d_width, d_height);		// SSW
		directions[10] = findHazardKernel(x, y, -1, 1, d_pixBinaryMap, d_width, d_height);		// SW
		directions[11] = findHazardKernel(x, y, -2, 1, d_pixBinaryMap, d_width, d_height);		// WSW

		directions[12] = findHazardKernel(x, y, -1, 0, d_pixBinaryMap, d_width, d_height);		// W

		directions[13] = findHazardKernel(x, y, -2, -1, d_pixBinaryMap, d_width, d_height);		// WNW
		directions[14] = findHazardKernel(x, y, -1, -1, d_pixBinaryMap, d_width, d_height);		// NW
		directions[15] = findHazardKernel(x, y, -1, -2, d_pixBinaryMap, d_width, d_height);		// NNW

		min_element = directions[0];
		for (int a = 1; a < compassDirections; a++)
		{
			if (directions[a] < min_element)
			{
				min_element = directions[a];
			}
			else
			{
				continue;
			}
		}

		d_pixHazardMap[x + y*d_width] = min_element * 5;
	}
}

int main(int argc, char* argv[])
{
	if (argc != 2)
	{
		cout << "\n # Usage: " << argv[0] << " <filename>\n" << endl;

		cout << " > Press [ENTER] to end the program..." << endl;
		getchar();

		return 1;
	}
	else
	{
		//
		execute.startTimer("EXECUTION TIMER");

		cout << "\n # HONS #GPU COMPONENT \n" << endl;

		//
		//rt.startTimer("Image Input");

		string filename = "images/";
		filename += argv[1];

		if (loadImage(filename))
		{

			//rt.endTimer("Image Input");

			//
			//rt.startTimer("Create Binary and Number of Hazard Maps");

			create_pixBinaryMap();
			pixHazardMap = new int[width*height];

			//rt.endTimer("Create Binary and Number of Hazard Maps");

			// //
			rt.startTimer("CUDA Algorithm");

			cout << " # RUNNING ALGORITHM \n" << endl;
			cudaWrapper();
			cout << "\n ## CUDA Wrapper Complete!\n" << endl;

			rt.endTimer("CUDA Algorithm");

			//
			//rt.startTimer("Image Output");

			cout << " # SAVING HAZARD IMAGE \n" << endl;
			saveHazardImage(filename);

			//rt.endTimer("Image Output");

			execute.endTimer("EXECUTION TIMER");

			cout << "\n > Press [ENTER] to end the program..." << endl;
			getchar();

			return 0;
		}
		else
		{
			cout << "\n > Press [ENTER] to end the program..." << endl;
			getchar();

			return 1;
		}
	}
}

hipError_t cudaWrapper()
{
	bool	*d_pixBinaryMap;
	int		*d_pixHazardMap;
	int		*d_width, *d_height;
	dim3 dimGrid(128, 128);
	dim3 dimBlock(32, 32);
	hipError_t cudaStatus;

	//pixHazardMap = new int[width*height];

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, " X hipSetDevice failed!");
		goto Error;
	}
	else
	{
		printf(" + hipSetDevice success!\n");
	}

	// // TIMER START
	rt.startTimer("CUDA Memory Allocation");

	cudaStatus = hipMalloc(&d_width, sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, " X hipMalloc d_width failed!");
		goto Error;
	}
	else
	{
		printf(" + hipMalloc d_width success!\n");
	}

	cudaStatus = hipMalloc(&d_height, sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, " X hipMalloc d_height failed!");
		goto Error;
	}
	else
	{
		printf(" + hipMalloc d_height success!\n");
	}

	cudaStatus = hipMalloc(&d_pixBinaryMap, width*height*sizeof(bool));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, " X hipMalloc d_pixBinaryMap failed!");
		goto Error;
	}
	else
	{
		printf(" + hipMalloc d_pixBinaryMap success!\n");
	}

	cudaStatus = hipMalloc(&d_pixHazardMap, width*height*sizeof(int));
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, " X hipMalloc d_pixHazardMap failed!");
		goto Error;
	}
	else
	{
		printf(" + hipMalloc d_pixHazardMap success!\n");
	}

	rt.endTimer("CUDA Memory Allocation");

	// // TIMER START
	rt.startTimer("CUDA Memory Copy - Host To Device");

	cudaStatus = hipMemcpy(d_width, &width, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, " X hipMemcpy d_width failed!");
		goto Error;
	}
	else
	{
		printf(" + hipMemcpy d_width success!\n");
	}

	cudaStatus = hipMemcpy(d_height, &height, sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, " X hipMemcpy d_height failed!");
		goto Error;
	}
	else
	{
		printf(" + hipMemcpy d_height success!\n");
	}

	cudaStatus = hipMemcpy(d_pixBinaryMap, pixBinaryMap, width*height*sizeof(bool), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, " X hipMemcpy d_pixBinaryMap failed!");
		goto Error;
	}
	else
	{
		printf(" + hipMemcpy d_pixBinaryMap success!\n");
	}

	cudaStatus = hipMemcpy(d_pixHazardMap, pixHazardMap, width*height*sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, " X hipMemcpy d_pixHazardMap failed! \t %s \t %d \n", hipGetErrorString(cudaStatus), hipGetErrorName(cudaStatus));
		goto Error;
	}
	else
	{
		printf(" + hipMemcpy d_pixHazardMap success!\n");
	}

	rt.endTimer("CUDA Memory Copy - Host To Device");

	// //  TIMER START
	rt.startTimer("Kernel Launching");

	kernel << < dimGrid, dimBlock >> >(d_pixBinaryMap, d_pixHazardMap, width, height);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, " X kernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	else
	{
		printf(" + kernel launch success!\n");
	}

	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, " X hipDeviceSynchronize returned error code %d after launching kernel!\n", cudaStatus);
		goto Error;
	}
	else
	{
		printf(" + hipDeviceSynchronize success!\n");
	}

	rt.endTimer("Kernel Launching");

	// // TIMER START
	rt.startTimer("CUDA Memory Copy - Device To Host");

	cudaStatus = hipMemcpy(pixHazardMap, d_pixHazardMap, width*height*sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess)
	{
		fprintf(stderr, " X hipMemcpy d_pixHazardMap return failed! \t %s \t %d \n", hipGetErrorString(cudaStatus), hipGetErrorName(cudaStatus));
		goto Error;
	}
	else
	{
		printf(" + hipMemcpy d_pixHazardMap return success!\n");
	}

	rt.endTimer("CUDA Memory Copy - Device To Host");

Error:
	hipFree(d_pixBinaryMap);
	hipFree(d_pixHazardMap);
	hipFree(d_width);
	hipFree(d_height);

	return cudaStatus;
}

bool loadImage(string filename)
{
	filename += ".ppm";
	string head, wid, ht, colrs;
	string r, g, b;
	ifstream in;

	in.open(filename);

	if (in.is_open())
	{
		cout << " - " << filename << " is open" << endl;

		getline(in, head);
		header = head;
		cout << " - magic header: " << head << '\n';

		getline(in, wid);
		width = atoi(wid.c_str());
		cout << " - width px: " << width;

		getline(in, ht);
		height = atoi(ht.c_str());
		cout << " height px: " << height << '\n';

		getline(in, colrs);
		colourRange = atoi(colrs.c_str());
		cout << " - colour range: " << colrs << '\n';

		pixArray = new pixVals[width*height];

		for (int j = 0; !in.eof() && j < width*height; j++)
		{
			getline(in, r);
			pixArray[j].R = atoi(r.c_str());
			getline(in, g);
			pixArray[j].G = atoi(g.c_str());
			getline(in, b);
			pixArray[j].B = atoi(b.c_str());
		}

		cout << endl;
		in.close();
		return true;
	}
	else
	{
		cerr << " ## ERROR ## Cannot open file: " << filename << "\n\n" << endl;
		return false;
	}
}

void create_pixBinaryMap()
{
	pixBinaryMap = new bool[width*height];

	for (int i = 0; i < width*height; i++)
	{
		if (pixArray[i].R == colourRange)
		{
			pixBinaryMap[i] = true;
		}
		else if (pixArray[i].G == colourRange)
		{
			pixBinaryMap[i] = false;
		}
	}
	cout << " ## Binary Map created!" << endl;
}

void saveHazardImage(string filename)
{
	filename += "_OUT.ppm";
	ofstream fout(filename);
	cout << " ## File output: " << filename << endl;

	fout << "P3\n";
	fout << width << "\n";
	fout << height << "\n";
	fout << "255\n";

	for (unsigned int i = 0; i < width*height; i++)
	{
		if (pixBinaryMap[i])
		{
			fout << "100" << "\n";
			fout << "0" << "\n";
			fout << "0" << "\n";
		}
		else if (pixHazardMap[i] >= 255)
		{
			fout << "255" << "\n";
			fout << "255" << "\n";
			fout << "255" << "\n";
		}
		else
		{
			fout << pixHazardMap[i] << "\n";
			fout << pixHazardMap[i] << "\n";
			fout << pixHazardMap[i] << "\n";
		}
	}

	fout << flush;
	fout.close();

	cout << "\n ## Output file '" << filename << "' saved!" << endl;
}
